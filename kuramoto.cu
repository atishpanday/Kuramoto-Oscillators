#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define h 0.01

#define lambda 0.001

__global__ void kuramoto(int *A, double *theta, double *w, double *k, double *prevk, int N, int iter, double adder) {
	int id = threadIdx.x;
	
	k[id] = w[id];
	
	for(int i = 0; i < N; i++) {
		k[id] += lambda * A[i*N + id] * sin(theta[iter*N + i] - (theta[iter*N + id] + (adder * prevk[i])));
	}
}

__global__ void update_theta(double *theta, double *k1, double *k2, double *k3, double *k4, int iter, int N) {
	int id = threadIdx.x;
	theta[(iter+1)*N + id] = theta[iter*N + id] + (double)(h * (k1[id] + 2*k2[id] + 2*k3[id] + k4[id])) / 6;
				
	while(theta[(iter+1)*N + id] > M_PI) {
		theta[(iter+1)*N + id] = 2 * M_PI - theta[(iter+1)*N + id];
	}
	
	while(theta[(iter+1)*N + id] < -M_PI) {
		theta[(iter+1)*N + id] += 2*M_PI;
	}
}

int main() {
	FILE *fptr_1, *fptr_2, *fptr_3;
	
	fptr_1 = fopen("./adjacency.txt", "r");
	fptr_2 = fopen("./initial_phase.txt", "r");
	fptr_3 = fopen("./omega.txt", "r");
	
	double *theta, *dtheta;
	double *omega, *domega;
	double *k0, *dk0, *dk1, *dk2, *dk3, *dk4;
	double *r;
	
	double c = 0, s = 0;
	
	int N = 10;

	int *A, *dA;
	
	A = (int *)malloc(N * N * sizeof(int));
	hipMalloc(&dA, N * N * sizeof(int));
	
	theta = (double *)malloc(N * 10000 * sizeof(double));
	hipMalloc(&dtheta, N * 10000 * sizeof(double));
	
	omega = (double*)malloc(N * sizeof(double));
	hipMalloc(&domega, N * sizeof(double));
	
	r = (double*)malloc(10000 * sizeof(double));
	
	for(int i = 0; i < N; i++) {
		c += cos(theta[i]);
		s += sin(theta[i]);
	}
	r[0] = sqrt(c*c + s*s) / N;
	c = 0;
	s = 0;
	
	hipMalloc(&dk0, N*sizeof(double));
	hipMalloc(&dk1, N*sizeof(double));
	hipMalloc(&dk2, N*sizeof(double));
	hipMalloc(&dk3, N*sizeof(double));
	hipMalloc(&dk4, N*sizeof(double));
	
	k0 = (double *)malloc(N * sizeof(double));
	for(int i = 0; i < N; i++) k0[i] = 0;
	hipMemcpy(dk0, k0, N * sizeof(double), hipMemcpyHostToDevice);
	
	for(int i = 0; i < N; i++) {
		fscanf(fptr_2, "%lf", &theta[i]);
		printf("%lf ", theta[i]);
	}
	hipMemcpy(dtheta, theta, N * 10000 * sizeof(double), hipMemcpyHostToDevice);
	printf("\n");
	
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			fscanf(fptr_1, "%d", &A[i*N + j]);
			printf("%d ", A[i*N + j]);
		}
		printf("\n");
	}
	hipMemcpy(dA, A, N * N * sizeof(int), hipMemcpyHostToDevice);
	
	for(int i = 0; i < N; i++) {
		fscanf(fptr_3, "%lf", &omega[i]);
		printf("%lf ", omega[i]);
	}
	hipMemcpy(domega, omega, N * sizeof(double), hipMemcpyHostToDevice);
	
	printf("\n");
	float t = 0;
	int iter = 0;
	
	while(t < 100) {
		
		kuramoto<<<1, N>>>(dA, dtheta, domega, dk1, dk0, N, iter, 0);
		hipDeviceSynchronize();
		
		kuramoto<<<1, N>>>(dA, dtheta, domega, dk2, dk1, N, iter, h/2);
		hipDeviceSynchronize();
		
		kuramoto<<<1, N>>>(dA, dtheta, domega, dk3, dk2, N, iter, h/2);
		hipDeviceSynchronize();
		
		kuramoto<<<1, N>>>(dA, dtheta, domega, dk4, dk3, N, iter, h);
		hipDeviceSynchronize();
		
		update_theta<<<1, N>>>(dtheta, dk1, dk2, dk3, dk4, iter, N);
		hipDeviceSynchronize();
		
		t += h;
		
		iter++;
		
		for(int i = 0; i < N; i++) {
			c += cos(theta[iter*N + i]);
			s += sin(theta[iter*N + i]);
		}
		r[iter] = sqrt(c*c + s*s) / N;
		c = 0;
		s = 0;
	}
	
	hipMemcpy(theta, dtheta, N * 10000 * sizeof(double), hipMemcpyDeviceToHost);
	
	for(int i = 0; i < N; i++) {
		printf("%lf ", theta[9900 + i]);
	}
	
	FILE *fptr_4;
	
	fptr_4 = fopen("./order_par.txt", "w");
	
	for(int i = 0; i < 10000; i++) {
		fprintf(fptr_4, "%lf ", r[i]);
	}
	
	printf("\n");
	
	fclose(fptr_1);
	fclose(fptr_2);
	fclose(fptr_3);
	fclose(fptr_4);
	
	free(A);
	free(theta);
	free(omega);
	free(r);
	free(k0);
	
	hipFree(dA);
	hipFree(dtheta);
	hipFree(domega);
	hipFree(dk0);
	hipFree(dk1);
	hipFree(dk2);
	hipFree(dk3);
	hipFree(dk4);
	return 0;
}


