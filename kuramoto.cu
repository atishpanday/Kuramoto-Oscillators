#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define h 0.01

#define lambda 0.5

__global__ void kuramoto(int *A, double *theta, double *w, 
				double *k, double *prevk, int N, int iter, double adder) {
	int id = threadIdx.x;
	
	theta[id] = w[id];
	
	for(int i = 0; i < N; i++) {
		k[id] += lambda * A[i*N + id] * sin((theta[i] + adder * prevk[i]) - theta[id + iter*N]);
	}
}

__global__ void update_theta(double *theta, double *k1, double *k2, double *k3, double *k4, int iter, int N) {
	int id = threadIdx.x;
	theta[(iter+1)*N + id] = theta[iter*N + id] + (double)(h * (k1[id] + 2*k2[id] + 2*k3[id] + k4[id])) / 6;
}

int main() {
	FILE *fptr_1, *fptr_2, *fptr_3;

	int N = 0;
	
	fptr_1 = fopen("./adjacency.txt", "r");
	fptr_2 = fopen("./initial_phase.txt", "r");
	fptr_3 = fopen("./omega.txt", "r");
	
	double ph, ff;
	
	float t = 0;
	
	do {
		ff = fscanf(fptr_2, "%lf", &ph);
		N++;
	} while(ff != EOF);
	
	N--;
	
	printf("count = %d\n", N);
	
	double *theta, *dtheta;
	
	theta = (double *)malloc(N * 10000 * sizeof(double));
	hipMalloc(&dtheta, N * 10000 * sizeof(double));
	
	for(int i = 0; i < N; i++) {
		fscanf(fptr_2, "%lf", &theta[i]);
	}
	
	for(int i = 0; i < N; i++) {
		printf("%lf ", theta[i]);
	}
	
	hipMemcpy(dtheta, theta, N * 10000 * sizeof(double), hipMemcpyHostToDevice);
	
	int *A, *dA;
	A = (int *)malloc(N * N * sizeof(int));
	hipMalloc(&dA, N * N * sizeof(int));
	
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			fscanf(fptr_1, "%d", &A[i*N + j]);
		}
	}
	
	hipMemcpy(dA, A, N * N * sizeof(int), hipMemcpyHostToDevice);
	
	double *omega, *domega;
	
	omega = (double*) malloc(N * sizeof(double));
	hipMalloc(&domega, N * sizeof(double));
	
	for(int i = 0; i < N; i++) {
		fscanf(fptr_3, "%lf", &omega[i]);
	}
	
	hipMemcpy(domega, omega, N * sizeof(double), hipMemcpyHostToDevice);
	
	int iter = 0;
	
	double *dk1, *dk2, *dk3, *dk4;
	hipMalloc(&dk1, N*sizeof(double));
	hipMalloc(&dk2, N*sizeof(double));
	hipMalloc(&dk3, N*sizeof(double));
	hipMalloc(&dk4, N*sizeof(double));
	
	while(t < 100) {
		kuramoto<<<1, N>>>(dA, dtheta, domega, dk1, NULL, N, iter, 0);
		hipDeviceSynchronize();
		
		kuramoto<<<1, N>>>(dA, dtheta, domega, dk2, dk1, N, iter, h/2);
		hipDeviceSynchronize();
		
		kuramoto<<<1, N>>>(dA, dtheta, domega, dk3, dk2, N, iter, h/2);
		hipDeviceSynchronize();
		
		kuramoto<<<1, N>>>(dA, dtheta, domega, dk4, dk3, N, iter, h);
		hipDeviceSynchronize();
		
		update_theta<<<1, N>>>(dtheta, dk1, dk2, dk3, dk4, iter, N);
		hipDeviceSynchronize();
		
		t += h;
		
		iter++;
	}
	
	hipMemcpy(theta, dtheta, N * 10000 * sizeof(double), hipMemcpyDeviceToHost);
	
	for(int i = 0; i < N; i++) {
		printf("%lf ", theta[i]);
	}
	
	fclose(fptr_1);
	fclose(fptr_2);
	fclose(fptr_3);
	
	return 0;
}


